#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "kernels.h"

__global__ void matmulKernel(float* A, float* B, float* C, int rowA, int colA, int rowB, int colB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float value = 0.0f;
    for(unsigned int stride = 0; stride < colA; stride += TILE_WIDTH) {
        // Shared memory for A and B
        __shared__ float As[TILE_WIDTH][TILE_WIDTH];
        __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

        // Load A and B into shared memory
        if (row < rowA && stride + threadIdx.x < colA) {
            As[threadIdx.y][threadIdx.x] = A[row * colA + stride + threadIdx.x];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }
        if (stride + threadIdx.y < colA && col < colB) {
            Bs[threadIdx.y][threadIdx.x] = B[(stride + threadIdx.y) * colB + col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        }
        __syncthreads();

        // Compute the value
        for (int k = 0; k < TILE_WIDTH; k++) {
            value += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }
        __syncthreads();
    }
    // Write the result to global memory
    if (row < rowA && col < colB) {
        C[row * colB + col] = value;
    }
}
// TODO - give option to use A, B if they are already on the GPU?
// TODO - To do so, pass already allocated pointers to the function
float* matmulGPU(float* A, float* B, int rowA, int colA, int rowB, int colB){
    assert(colA == rowB); // Ensure the matrices can be multiplied
    float* C = (float*)malloc(rowA * colB * sizeof(float));
    if (C == NULL) {
        fprintf(stderr, "Error allocating memory for result matrix\n");
        exit(EXIT_FAILURE);
    }
    // ++++++++++++ Allocate GPU float* matrices ++++++++++++
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, rowA * colA * sizeof(float));
    hipMalloc((void**)&d_B, rowB * colB * sizeof(float));
    hipMalloc((void**)&d_C, rowA * colB * sizeof(float));

    // ++++++++++++ Copy A and B to GPU ++++++++++++
    hipMemcpy(d_A, A, rowA * colA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, rowB * colB * sizeof(float), hipMemcpyHostToDevice);
    // ++++++++++++ Launch kernel ++++++++++++
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((colB + threadsPerBlock.x - 1) / threadsPerBlock.x, (rowA + threadsPerBlock.y - 1) / threadsPerBlock.y);
    matmulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, rowA, colA, rowB, colB);
    // ++++++++++++ Copy result back to CPU ++++++++++++
    hipMemcpy(C, d_C, rowA * colB * sizeof(float), hipMemcpyDeviceToHost);
    // ++++++++++++ Free GPU memory ++++++++++++
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return C;
}


float* matmulCPU(float* A, float* B, int rowA, int colA, int rowB, int colB) {
    assert(colA == rowB); // Ensure the matrices can be multiplied
    float* C = (float*)malloc(rowA * colB * sizeof(float));
    if (C == NULL) {
        fprintf(stderr, "Error allocating memory for result matrix\n");
        exit(EXIT_FAILURE);
    }
    for (int i = 0; i < rowA; i++) {
        for (int j = 0; j < colB; j++) {
            C[i * colB + j] = 0;
            for (int k = 0; k < colA; k++) {
                C[i * colB + j] += A[i * colA + k] * B[k * colB + j];
            }
        }
    }
    return C;
}

// M rows, N columns: MxN
// TODO make reductions over columns dimension
// TODO input[row*N+col] inside the code seems to need optimization - coalesing? shared? - also for output
// from: https://github.com/vectorquantized/100daysofcuda/blob/main/src/day_7/online_softmax.cu
__global__ void softmaxKernel(float* input, float* output, int M, int N) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (row < M){
        float max_val = -__FLT_MAX__;
        float norm = 0;

        for (int col = 0; col < N; col++){
            float val = input[row * N + col];
            if(val > max_val){
                norm *= expf(max_val - val);
                max_val = val;
            }
            norm += expf(val - max_val);
        }

        for (int col = 0; col < N; col++){
            output[row * N + col] = expf(input[row * N + col] - max_val) / (norm + EPSILON);
        }
    }
}

float* online_softmax(float* input, int M, int N) {
    float* output = (float*) malloc(M * N * sizeof(float));
    if (output == NULL){
        fprintf(stderr, "Error allocating memory for result matrix\n");
        exit(EXIT_FAILURE);
    }
    // +++++++++++++ Allocate GPU float* matrices +++++++++
    float *d_input, *d_output;
    hipMalloc((void**)&d_input, M * N * sizeof(float));
    hipMalloc((void**)&d_output, M * N * sizeof(float));

    hipMemcpy(d_input, input, M * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(256);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / threadsPerBlock.x);
    softmaxKernel<<<numBlocks, threadsPerBlock>>>(d_input, d_output, M, N);
    hipMemcpy(output, d_output, M * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return output;
}
