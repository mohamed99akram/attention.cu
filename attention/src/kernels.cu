#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "kernels.h"

__global__ void matmulKernel(float* A, float* B, float* C, int rowA, int colA, int rowB, int colB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float value = 0.0f;
    for(unsigned int stride = 0; stride < colA; stride += TILE_WIDTH) {
        // Shared memory for A and B
        __shared__ float As[TILE_WIDTH][TILE_WIDTH];
        __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

        // Load A and B into shared memory
        if (row < rowA && stride + threadIdx.x < colA) {
            As[threadIdx.y][threadIdx.x] = A[row * colA + stride + threadIdx.x];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }
        if (stride + threadIdx.y < colA && col < colB) {
            Bs[threadIdx.y][threadIdx.x] = B[(stride + threadIdx.y) * colB + col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        }
        __syncthreads();

        // Compute the value
        for (int k = 0; k < TILE_WIDTH; k++) {
            value += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }
        __syncthreads();
    }
    // Write the result to global memory
    if (row < rowA && col < colB) {
        C[row * colB + col] = value;
    }
}
// TODO - give option to use A, B if they are already on the GPU?
// TODO - To do so, pass already allocated pointers to the function
float* matmulGPU(float* A, float* B, int rowA, int colA, int rowB, int colB){
    assert(colA == rowB); // Ensure the matrices can be multiplied
    float* C = (float*)malloc(rowA * colB * sizeof(float));
    if (C == NULL) {
        fprintf(stderr, "Error allocating memory for result matrix\n");
        exit(EXIT_FAILURE);
    }
    // ++++++++++++ Allocate GPU float* matrices ++++++++++++
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, rowA * colA * sizeof(float));
    hipMalloc((void**)&d_B, rowB * colB * sizeof(float));
    hipMalloc((void**)&d_C, rowA * colB * sizeof(float));

    // ++++++++++++ Copy A and B to GPU ++++++++++++
    hipMemcpy(d_A, A, rowA * colA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, rowB * colB * sizeof(float), hipMemcpyHostToDevice);
    // ++++++++++++ Launch kernel ++++++++++++
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((colB + threadsPerBlock.x - 1) / threadsPerBlock.x, (rowA + threadsPerBlock.y - 1) / threadsPerBlock.y);
    matmulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, rowA, colA, rowB, colB);
    // ++++++++++++ Copy result back to CPU ++++++++++++
    hipMemcpy(C, d_C, rowA * colB * sizeof(float), hipMemcpyDeviceToHost);
    // ++++++++++++ Free GPU memory ++++++++++++
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return C;
}


float* matmulCPU(float* A, float* B, int rowA, int colA, int rowB, int colB) {
    assert(colA == rowB); // Ensure the matrices can be multiplied
    float* C = (float*)malloc(rowA * colB * sizeof(float));
    if (C == NULL) {
        fprintf(stderr, "Error allocating memory for result matrix\n");
        exit(EXIT_FAILURE);
    }
    for (int i = 0; i < rowA; i++) {
        for (int j = 0; j < colB; j++) {
            C[i * colB + j] = 0;
            for (int k = 0; k < colA; k++) {
                C[i * colB + j] += A[i * colA + k] * B[k * colB + j];
            }
        }
    }
    return C;
}

// M rows, N columns: MxN
// TODO make reductions over columns dimension
// TODO input[row*N+col] inside the code seems to need optimization - coalesing? shared? - also for output
// inspired from: https://github.com/vectorquantized/100daysofcuda/blob/main/src/day_7/online_softmax.cu
__global__ void softmaxKernel(float* input, float* output, int M, int N) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (row < M){
        float max_val = -__FLT_MAX__;
        float norm = 0;

        for (int col = 0; col < N; col++){
            float val = input[row * N + col];
            if(val > max_val){
                norm *= expf(max_val - val);
                max_val = val;
            }
            norm += expf(val - max_val);
        }

        for (int col = 0; col < N; col++){
            output[row * N + col] = expf(input[row * N + col] - max_val) / (norm + EPSILON);
        }
    }
}

float* online_softmax(float* input, int M, int N) {
    float* output = (float*) malloc(M * N * sizeof(float));
    if (output == NULL){
        fprintf(stderr, "Error allocating memory for result matrix\n");
        exit(EXIT_FAILURE);
    }
    // +++++++++++++ Allocate GPU float* matrices +++++++++
    float *d_input, *d_output;
    hipMalloc((void**)&d_input, M * N * sizeof(float));
    hipMalloc((void**)&d_output, M * N * sizeof(float));

    hipMemcpy(d_input, input, M * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(256);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / threadsPerBlock.x);
    softmaxKernel<<<numBlocks, threadsPerBlock>>>(d_input, d_output, M, N);
    hipMemcpy(output, d_output, M * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return output;
}

// inspired from https://github.com/vectorquantized/100daysofcuda/blob/main/src/day_10/self_attention.cu
// TODO may need to consider torch-like movement-free transpose
// input: MxN, output: NxM
__global__ void tiled_transposeKernel(float* input, float* output, int M, int N){
    __shared__ float temp[TILE_WIDTH][TILE_WIDTH]; // TODO try TILE_WIDTH+1 & compare

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    if (row < M && col < N){
        temp[threadIdx.y][threadIdx.x] = input[row * N + col];
    }
    __syncthreads();

    int row_t = col;
    int col_t = row;
    int M_t = N;
    int N_t = M;
    if(row_t < M_t && col_t < N_t){
        output[row_t * N_t + col_t] = temp[threadIdx.y][threadIdx.x];
    }
}
float* tiled_tranpose(float* input, int M, int N){
    float* output = (float*) malloc(M*N*sizeof(float));
    if (output == NULL){
        fprintf(stderr, "Error allocating memory for result matrix\n");
        exit(EXIT_FAILURE);
    }
    float *d_input, *d_output;
    hipMalloc((void**)&d_input, M*N*sizeof(float));
    hipMalloc((void**)&d_output, M*N*sizeof(float));

    hipMemcpy(d_input, input, M * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, ( M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    tiled_transposeKernel<<<numBlocks, threadsPerBlock>>>(d_input, d_output, M, N);
    hipMemcpy(output, d_output, M * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    return output;
}

/***********     SELF ATTENTION     *************/
/*
Q.shape: (L, d_k)
K.shape: (L, d_k)
V.shape: (L, d_k)

L: sequence length
d_k = d_model/h
Taking CPU matrices - Putting on GPU then computing & returning result on CPU
*/
__global__ void mul(float* A, float* B, float num, int size){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < size){
        B[tid] = A[tid]*num;
    }
}
// float* self_attentionGPU(float* Q, float* K, float* V, int L, int d_k){
    
// }
float* self_attention(float* Q, float* K, float* V, int L, int d_k){
    float* output = (float*)malloc(L * d_k * sizeof(float));
    if (output == NULL){
        fprintf(stderr, "Error allocating memory for result matrix\n");
        exit(EXIT_FAILURE);
    }
    float* d_output; // L x d_v 
    float *d_Q; // L x d_k
    float *d_K; // L x d_k
    float *d_V; // L x d_k 
    float *d_KT; // d_k x L -- Output of K.T
    float *d_QKT; // L x L -- ouput of Q@K.T
    float *d_QKT2; // L x L -- Output of Q@K.T/sqrt(d_k)
    float *d_S; // L x L -- Output of Softmax(Q@K.T/sqrt(d_k))

    // Allocate GPU memory
    hipMalloc((void**)&d_output, L * d_k * sizeof(float));
    hipMalloc((void**)&d_Q, L * d_k * sizeof(float));
    hipMalloc((void**)&d_K, L * d_k * sizeof(float));
    hipMalloc((void**)&d_V, L * d_k * sizeof(float));
    hipMalloc((void**)&d_KT, d_k * L * sizeof(float));
    hipMalloc((void**)&d_QKT, L * L * sizeof(float));
    hipMalloc((void**)&d_QKT2, L * L * sizeof(float));
    hipMalloc((void**)&d_S, L * L * sizeof(float));

    // Copy inputs (Q, K, V)
    hipMemcpy(d_Q, Q, L * d_k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_K, K, L * d_k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V, V, L * d_k * sizeof(float), hipMemcpyHostToDevice);

    // Take Transpose of K
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks1((d_k + threadsPerBlock.x - 1) / threadsPerBlock.x, ( L + threadsPerBlock.y - 1) / threadsPerBlock.y);

    tiled_transposeKernel<<<numBlocks1, threadsPerBlock>>>(d_K, d_KT, L, d_k);

    // Compute Q@K.T
    dim3 numBlocks2((L + threadsPerBlock.x - 1) / threadsPerBlock.x, (L + threadsPerBlock.y - 1) / threadsPerBlock.y);
    matmulKernel<<<numBlocks2, threadsPerBlock>>>(d_Q, d_KT, d_QKT, L, d_k, d_k, L);
    
    // Compute Q@K.T / sqrt(d_k)
    float rec_sq_dk = 1 / sqrt(d_k); // reciprocal of square root of d_k
    
    dim3 threadsPerBlock3(256);
    dim3 numBlocks3((L * L + threadsPerBlock3.x - 1) / threadsPerBlock3.x);
    mul<<<numBlocks3, threadsPerBlock3>>>(d_QKT, d_QKT2, rec_sq_dk, L * L);

    // Compute Softmax(Q@K.T/sqrt(d_k))
    dim3 threadsPerBlock4(256);
    dim3 numBlocks4((L + threadsPerBlock.x - 1) / threadsPerBlock.x);
    softmaxKernel<<<numBlocks4, threadsPerBlock4>>>(d_QKT2, d_S, L, L);

    // Compute S@V
    dim3 threadsPerBlock5(16, 16);
    dim3 numBlocks5((d_k + threadsPerBlock.x - 1) / threadsPerBlock.x, (L + threadsPerBlock.y - 1) / threadsPerBlock.y);
    matmulKernel<<<numBlocks5, threadsPerBlock5>>>(d_S, d_V, d_output, L, L, L, d_k);

    hipMemcpy(output, d_output, L * d_k * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_output);
    hipFree(d_Q);
    hipFree(d_K);
    hipFree(d_V);
    hipFree(d_KT);
    hipFree(d_QKT);
    hipFree(d_QKT2);
    hipFree(d_S);

    return output;
}